//
// Wrapper for cublasSgemm function.
//
// Alan Gray, NVIDIA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

bool alreadyAllocated_sgemm = false;
bool alreadyAllocated_sgemm_handle = false;

float **d_Aarray_sgemm;
float **d_Barray_sgemm;
float **d_Carray_sgemm;

float **Aarray_sgemm;
float **Barray_sgemm;
float **Carray_sgemm;

hipblasHandle_t handle_sgemm;

extern "C" void cublasSgemmStridedBatched_wrapper(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, long long tda, const float *B, int ldb, long long tdb, float beta, float *C, int ldc, long long tdc, int batchCount)
{

  printf("CUBLAS m=%d,n=%d,k=%d,batchcount=%d\n", m, n, k, batchCount);

  hipblasOperation_t op_t1 = HIPBLAS_OP_N, op_t2 = HIPBLAS_OP_N;

  if (transa == 'T' || transa == 't')
    op_t1 = HIPBLAS_OP_T;

  if (transb == 'T' || transb == 't')
    op_t2 = HIPBLAS_OP_T;

  if (!alreadyAllocated_sgemm_handle)
  {
    hipblasCreate(&handle_sgemm);
    alreadyAllocated_sgemm_handle = true;
  }
  printf("before sgemm: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  if (hipblasSgemmStridedBatched(handle_sgemm, op_t1, op_t2, m, n, k, &alpha, (const float *)A, lda, tda, (const float *)B, ldb, tdb, &beta, (float *)C, ldc, tdc, batchCount) == HIPBLAS_STATUS_SUCCESS)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipblasSgemmBatched succeeded\n");
  }
  printf("after sgemm: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
}

extern "C" void cublasSgemmBatched_finalize()
{

  if (alreadyAllocated_sgemm)
  {

    hipFree(Aarray_sgemm);
    hipFree(Barray_sgemm);
    hipFree(Carray_sgemm);

    hipFree(d_Aarray_sgemm);
    hipFree(d_Barray_sgemm);
    hipFree(d_Carray_sgemm);
  }

  if (alreadyAllocated_sgemm_handle)
  {
    hipblasDestroy(handle_sgemm);
  }
}