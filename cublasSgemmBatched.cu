//
// Wrapper for cublasSgemm function.
//
// Alan Gray, NVIDIA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

bool alreadyAllocated_sgemm = false;
bool alreadyAllocated_sgemm_handle = false;

float **d_Aarray_sgemm;
float **d_Barray_sgemm;
float **d_Carray_sgemm;

float **Aarray_sgemm;
float **Barray_sgemm;
float **Carray_sgemm;

hipblasHandle_t handle_sgemm;

extern "C" void cublasSgemmBatched_wrapper(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, int tda, const float *B, int ldb, int tdb, float beta, float *C, int ldc, int tdc, int batchCount)
{

  printf("CUBLAS m=%d,n=%d,k=%d,batchcount=%d\n", m, n, k, batchCount);

  hipblasOperation_t op_t1 = HIPBLAS_OP_N, op_t2 = HIPBLAS_OP_N;

  if (transa == 'T' || transa == 't')
    op_t1 = HIPBLAS_OP_T;

  if (transb == 'T' || transb == 't')
    op_t2 = HIPBLAS_OP_T;

  //float **Aarray_sgemm = (float**) malloc(batchCount*sizeof(float*));
  //float **Barray_sgemm = (float**) malloc(batchCount*sizeof(float*));
  //float **Carray_sgemm = (float**) malloc(batchCount*sizeof(float*));

  if (!alreadyAllocated_sgemm_handle)
  {
    if (hipblasCreate(&handle_sgemm) == HIPBLAS_STATUS_SUCCESS)
    {
      printf("Cuda in cublasSgemmBatched.cu: hipblasCreate succeeded\n");
    }
    printf("after cublasCreate\n");
    printf("after cublasCreate: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
    alreadyAllocated_sgemm_handle = true;
  }

  if (!alreadyAllocated_sgemm)
  {
    if (hipHostMalloc(&Aarray_sgemm, batchCount * sizeof(float *), hipHostMallocDefault) == hipSuccess)
    {
      printf("Cuda in cublasSgemmBatched.cu: hipHostMalloc A succeeded\n");
    }
    printf("after hipHostMalloc A\n");
    printf("after hipHostMalloc A: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
    if (hipHostMalloc(&Barray_sgemm, batchCount * sizeof(float *), hipHostMallocDefault) == hipSuccess)
    {
      printf("Cuda in cublasSgemmBatched.cu: hipHostMalloc B succeeded\n");
    }
    printf("after hipHostMalloc B\n");
    printf("after hipHostMalloc B: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
    if (hipHostMalloc(&Carray_sgemm, batchCount * sizeof(float *), hipHostMallocDefault) == hipSuccess)
    {
      printf("Cuda in cublasSgemmBatched.cu: hipHostMalloc C succeeded\n");
    }
    printf("after hipHostMalloc C\n");
    printf("after hipHostMalloc C: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
    alreadyAllocated_sgemm = true;
  }

  if (hipMalloc(&d_Aarray_sgemm, batchCount * sizeof(float *)) == hipSuccess)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipMalloc A succeeded\n");
  }
  printf("after hipMalloc A\n");
  printf("after hipMalloc A: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  if (hipMalloc(&d_Barray_sgemm, batchCount * sizeof(float *)) == hipSuccess)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipMalloc A succeeded\n");
  }
  printf("after hipMalloc B\n");
  printf("after hipMalloc B: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  if (hipMalloc(&d_Carray_sgemm, batchCount * sizeof(float *)) == hipSuccess)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipMalloc A succeeded\n");
  }
  printf("after hipMalloc C\n");
  printf("after hipMalloc C: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  int i;
  for (i = 0; i < batchCount; i++)
  {
    Aarray_sgemm[i] = (float *)&(A[i * lda * tda]);
    Barray_sgemm[i] = (float *)&(B[i * ldb * tdb]);
    Carray_sgemm[i] = (float *)&(C[i * ldc * tdc]);
  }
  if (hipMemcpy(d_Aarray_sgemm, Aarray_sgemm, batchCount * sizeof(float *), hipMemcpyHostToDevice) == hipSuccess)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipMemcpy A succeeded\n");
  }
  printf("after hipMemcpy A\n");
  printf("after hipMemcpy A: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  if (hipMemcpy(d_Barray_sgemm, Barray_sgemm, batchCount * sizeof(float *), hipMemcpyHostToDevice) == hipSuccess)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipMemcpy B succeeded\n");
  }
  printf("after hipMemcpy B\n");
  printf("after hipMemcpy B: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  if (hipMemcpy(d_Carray_sgemm, Carray_sgemm, batchCount * sizeof(float *), hipMemcpyHostToDevice) == hipSuccess)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipMemcpy C succeeded\n");
  }
  printf("after hipMemcpy C\n");
  printf("before sgemm: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  if (hipblasSgemmBatched(handle_sgemm, op_t1, op_t2, m, n, k, &alpha, (const float **)d_Aarray_sgemm, lda, (const float **)d_Barray_sgemm, ldb, &beta, (float **)d_Carray_sgemm, ldc, batchCount) == HIPBLAS_STATUS_SUCCESS)
  {
    printf("Cuda in cublasSgemmBatched.cu: hipblasSgemmBatched succeeded\n");
  }
  printf("after sgemm: cudaDeviceSynchronize=%d\n", hipDeviceSynchronize());
  printf("after sgemm\n");
  int syncres = hipDeviceSynchronize();
  printf("cublasSgemmBatched.cu: cudaDeviceSynchronize() returns %d\n", syncres);
  //cudaDeviceSynchronize();
  if (syncres != hipSuccess)
  {
    fprintf(stderr, "Cuda error 2 in cublasSgemmBatched.cu: Failed to synchronize\n");
    return;
  }

  //cudaFree(Aarray_sgemm);
  //cudaFree(Barray_sgemm);
  //cudaFree(Carray_sgemm);

  hipFree(d_Aarray_sgemm);
  hipFree(d_Barray_sgemm);
  hipFree(d_Carray_sgemm);
  //cublasDestroy(handle_sgemm);
}

extern "C" void cublasSgemmStridedBatched_wrapper(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, long long tda, const float *B, int ldb, long long tdb, float beta, float *C, int ldc, long long tdc, int batchCount)
{

  // printf("CUBLAS m=%d,n=%d,k=%d,batchcount=%d\n",m,n,k,batchCount);

  hipblasOperation_t op_t1 = HIPBLAS_OP_N, op_t2 = HIPBLAS_OP_N;

  if (transa == 'T' || transa == 't')
    op_t1 = HIPBLAS_OP_T;

  if (transb == 'T' || transb == 't')
    op_t2 = HIPBLAS_OP_T;

  if (!alreadyAllocated_sgemm_handle)
  {
    hipblasCreate(&handle_sgemm);
    alreadyAllocated_sgemm_handle = true;
  }
  hipblasSgemmStridedBatched(handle_sgemm, op_t1, op_t2, m, n, k, &alpha, (const float *)A, lda, tda, (const float *)B, ldb, tdb, &beta, (float *)C, ldc, tdc, batchCount);
}

extern "C" void cublasSgemmBatched_finalize()
{

  if (alreadyAllocated_sgemm)
  {

    hipFree(Aarray_sgemm);
    hipFree(Barray_sgemm);
    hipFree(Carray_sgemm);

    hipFree(d_Aarray_sgemm);
    hipFree(d_Barray_sgemm);
    hipFree(d_Carray_sgemm);
  }

  if (alreadyAllocated_sgemm_handle)
  {
    hipblasDestroy(handle_sgemm);
  }
}